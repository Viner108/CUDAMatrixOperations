#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "kernel.h"



int main()
{
	float* hA;
	float* dA;

	float* hB;
	float* dB;

	float* hC;
	float* dC;

	int  N_thread = 3;
	int vectorSize = N_thread * 1;
	int matrixSize = vectorSize * vectorSize;
	int N_blocks;
	int i;
	int j;
	unsigned int matrixMem_size = sizeof(float) * matrixSize;

	hA = (float*)malloc(matrixMem_size);
	hB = (float*)malloc(matrixMem_size);
	hC = (float*)malloc(matrixMem_size);

	hipError_t err;

	err = hipMalloc((void**)&dA, matrixMem_size);
	if (err != hipSuccess) {
		fprintf(stderr, "Cannot allocate GPU memory: %s\n", hipGetErrorString(err));
		return 1;
	}

	err = hipMalloc((void**)&dB, matrixMem_size);
	if (err != hipSuccess) {
		fprintf(stderr, "Cannot allocate GPU memory: %s\n", hipGetErrorString(err));
		return 1;
	}

	err = hipMalloc((void**)&dC, matrixMem_size);
	if (err != hipSuccess) {
		fprintf(stderr, "Cannot allocate GPU memory: %s\n", hipGetErrorString(err));
		return 1;
	}

	for (i = 0; i < vectorSize; i++) {
		for (int j = 0; j < vectorSize; j++) {
			hA[i * vectorSize + j] = j+1;
			printf("A[%d,%d] = %.5f\n", i, j, hA[i * vectorSize + j]);
			hB[i * vectorSize + j] = j+1;
			printf("B[%d,%d] = %.5f\n", i, j, hB[i * vectorSize + j]);
			hC[i * vectorSize + j] = 0.0f;

		}
	}

	N_blocks = matrixSize / N_thread;

	hipMemcpy(dA, hA, matrixMem_size, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, matrixMem_size, hipMemcpyHostToDevice);

	function << < N_blocks, N_thread >> > (dA, dB, dC, vectorSize);

	hipMemcpy(hC, dC, matrixMem_size, hipMemcpyDeviceToHost);

	for (i = 0; i < vectorSize; i++) {
		for (int j = 0; j < vectorSize; j++) {
			printf("C[%d,%d] = %.5f\n", i, j, hC[i * vectorSize + j]);

		}
	}

	free(hA);
	free(hB);
	free(hC);

	hipFree(dA);
	hipFree(dB);
	hipFree(dC);


	return 0;

}
