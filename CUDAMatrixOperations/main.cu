#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "matrixXMatrix.h"
#include "matrixExponentiation.h"
#include <chrono>


int matrixMultiplication();
int matrixExponention(int exponent);

int main()
{
	return matrixExponention(5);
	//return matrixMultiplication();

}
int matrixExponention(int exponent) {

	float timerValueGPU, timerValueCPU;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	double* hA;
	double* dA;

	double* hB;
	double* dB;

	double* hC;
	double* dC;

	int  N_thread = 1000;
	int vectorSize = N_thread * 1;
	int matrixSize = vectorSize * vectorSize;
	int N_blocks;
	int i;
	int j;
	unsigned int matrixMem_size = sizeof(double) * matrixSize;

	hA = (double*)malloc(matrixMem_size);
	hB = (double*)malloc(matrixMem_size);
	hC = (double*)malloc(matrixMem_size);

	
	hipError_t err;

	err = hipMalloc((void**)&dA, matrixMem_size);
	if (err != hipSuccess) {
		fprintf(stderr, "Cannot allocate GPU memory: %s\n", hipGetErrorString(err));
		return 1;
	}

	err = hipMalloc((void**)&dB, matrixMem_size);
	if (err != hipSuccess) {
		fprintf(stderr, "Cannot allocate GPU memory: %s\n", hipGetErrorString(err));
		return 1;
	}

	err = hipMalloc((void**)&dC, matrixMem_size);
	if (err != hipSuccess) {
		fprintf(stderr, "Cannot allocate GPU memory: %s\n", hipGetErrorString(err));
		return 1;
	}

	for (i = 0; i < vectorSize; i++) {
		for (int j = 0; j < vectorSize; j++) {
			hA[i * vectorSize + j] = j+1;
			//printf("A[%d,%d] = %.5f\n", i, j, hA[i * vectorSize + j]);
			hB[i * vectorSize + j] = j+1;
			//printf("B[%d,%d] = %.5f\n", i, j, hB[i * vectorSize + j]);
			hC[i * vectorSize + j] = 0.0f;
		}
	}


	printf("\n");

	N_blocks = matrixSize / N_thread;
	hipEventRecord(start, 0);

	hipMemcpy(dA, hA, matrixMem_size, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, matrixMem_size, hipMemcpyHostToDevice);

	

	matrixExponentiation (dA, dB, dC, exponent, vectorSize);
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
		return 1;
	}
	

	err = hipMemcpy(hC, dC, matrixMem_size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "Cannot copy data device/host : %s\n", hipGetErrorString(err));
		return 1;
	}

	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timerValueGPU, start, stop);
	printf("\n GPU calculation time: %f ms\n", timerValueGPU);



	for (i = 0; i < vectorSize; i++) {
		for (int j = 0; j < vectorSize; j++) {
			printf("C[%d,%d] = %.5f\n", i, j, hC[i * vectorSize + j]);

		}
	}



	free(hA);
	free(hB);
	free(hC);

	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

}

int matrixMultiplication() {
	double* hA;
	double* dA;

	double* hB;
	double* dB;

	double* hC;
	double* dC;

	int  N_thread = 2;
	int vectorSize = N_thread * 1;
	int matrixSize = vectorSize * vectorSize;
	int N_blocks;
	int i;
	int j;
	unsigned int matrixMem_size = sizeof(double) * matrixSize;

	hA = (double*)malloc(matrixMem_size);
	hB = (double*)malloc(matrixMem_size);
	hC = (double*)malloc(matrixMem_size);

	hipError_t err;

	err = hipMalloc((void**)&dA, matrixMem_size);
	if (err != hipSuccess) {
		fprintf(stderr, "Cannot allocate GPU memory: %s\n", hipGetErrorString(err));
		return 1;
	}

	err = hipMalloc((void**)&dB, matrixMem_size);
	if (err != hipSuccess) {
		fprintf(stderr, "Cannot allocate GPU memory: %s\n", hipGetErrorString(err));
		return 1;
	}

	err = hipMalloc((void**)&dC, matrixMem_size);
	if (err != hipSuccess) {
		fprintf(stderr, "Cannot allocate GPU memory: %s\n", hipGetErrorString(err));
		return 1;
	}

	//hA[0] = 1;
	//hA[1] = 2;
	//hA[2] = 0;
	//hA[3] = 3;
	//hB[0] = 2;
	//hB[1] = 1;
	//hB[2] = 5;
	//hB[3] = 7;

	for (i = 0; i < vectorSize; i++) {
		for (int j = 0; j < vectorSize; j++) {
			hA[i * vectorSize + j] = j+1;
			//printf("A[%d,%d] = %.5f\n", i, j, hA[i * vectorSize + j]);
			hB[i * vectorSize + j] = j+1;
			//printf("B[%d,%d] = %.5f\n", i, j, hB[i * vectorSize + j]);
			hC[i * vectorSize + j] = 0.0f;

		}
	}

	printf("\n");

	N_blocks = matrixSize / N_thread;

	hipMemcpy(dA, hA, matrixMem_size, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, matrixMem_size, hipMemcpyHostToDevice);

	functionX << < N_blocks, N_thread >> > (dA, dB, dC, vectorSize);

	hipMemcpy(hC, dC, matrixMem_size, hipMemcpyDeviceToHost);

	for (i = 0; i < vectorSize; i++) {
		for (int j = 0; j < vectorSize; j++) {
			printf("C[%d,%d] = %.5f\n", i, j, hC[i * vectorSize + j]);

		}
	}

	free(hA);
	free(hB);
	free(hC);

	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

	return 0;
}
