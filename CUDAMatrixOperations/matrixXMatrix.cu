#include "hip/hip_runtime.h"
#include "matrixXMatrix.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>

__global__ void functionX(float* dA, float* dB, float* dC, int vectorSize)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < vectorSize*vectorSize) {
        int k = i / vectorSize;
        int n = i % vectorSize;
        float sum = 0.0f;
        for (int j = 0; j < vectorSize; j++) {
        sum += dA[k * vectorSize + j] * dB[j * vectorSize + n];
        }
        dC[i] = sum;
    }


}
