#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include "matrixXMatrix.h"

 void matrixExponentiation(float* dA, float* dB, float* dC, int exponent, int vectorSize)
{
	for (int i = 1; i < exponent; i++)
	{
		if (i == 1)
		{
			functionX << < vectorSize, vectorSize >> > (dA, dA, dC, vectorSize);
		}
		else {
			functionX << < vectorSize, vectorSize >> > (dC, dA, dC, vectorSize);
		}

	}	
}