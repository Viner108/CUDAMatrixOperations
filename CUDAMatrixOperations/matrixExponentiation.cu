#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include "matrixXMatrix.h"

 void matrixExponentiation(double* dA, double* dB, double* dC, int exponent, int vectorSize)
{
	for (int i = 1; i < exponent; i++)
	{
		if (i == 1)
		{
			functionX << < vectorSize, vectorSize >> > (dA, dA, dC, vectorSize);
		}
		else {
			functionX << < vectorSize, vectorSize >> > (dC, dA, dC, vectorSize);
		}

	}	
}