#include "hip/hip_runtime.h"
#include "kernel.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>

__global__ void function(float* dA, float* dB, float* dC, int vectorSize)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < vectorSize*vectorSize) {
        float sum = 0.0f;
        for (int j = 0; j < vectorSize; j++) {
            sum += dA[i * vectorSize + j] * dB[j * vectorSize + i];
        }
        dC[i] = sum;
    }


}