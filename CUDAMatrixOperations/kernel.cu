#include "hip/hip_runtime.h"
#include "kernel.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>

__global__ void function(float* dA, float* dB, float* dC, int vectorSize)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < vectorSize*vectorSize) {
        int n = i % vectorSize;
        float sum = 0.0f;
        for (int j = 0; j < vectorSize; j++) {
        sum += dA[n * vectorSize + j] * dB[j * vectorSize + n];
        }
        dC[i] = sum;
    }


}